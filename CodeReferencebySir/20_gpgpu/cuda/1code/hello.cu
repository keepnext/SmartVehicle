#include "hip/hip_runtime.h"
/*
*	Hello World! ��� Ŀ�� �Լ� �ǽ�
*/

////////////////////////////////////////////////////////////////////////////////

#include "../common.cuh"

__constant__ char share[] = "Hello CUDA!";

__device__ void CopyString(char *dest, const char *source)
{
	int i = 0;
	while (source[i] != '\0')
	{
		dest[i] = source[i];
		i++;
	}
	dest[i] = '\0';
}

__global__ void Kernel_Hello(char *arg)
{
	printf("\n");
	printf("We arrived in Kernel: %s\n", share);

	arg[0] = 'H';
	arg[1] = 'e';
	arg[2] = 'l';
	arg[3] = 'l';
	arg[4] = 'o';
	arg[5] = ',';
	arg[6] = ' ';
	arg[7] = 'W';
	arg[8] = 'o';
	arg[9] = 'r';
	arg[10] = 'l';
	arg[11] = 'd';
	arg[12] = '!';
	arg[13] = '\0';
	
	printf("Kernel is writing \"Hello, World!\" ...");

	const char *buff = "Good-bye CUDA!";
	CopyString(share, buff);

	printf("\n");
	printf("Now we are leaving Kernel: %s\n", share);
}

// [To do1]
// Ŀ�� �Լ� hello2(__global__, int arg)�� �߰��ϰ�
// Host �ڵ忡�� hello2 Ŀ���� ȣ���ϼ���
// �׸��� ���� arg���� ȭ�鿡 ����ϼ���
__global__ void Kernel_Hello2(int arg)
{
	printf("hello2 in Kernel: %s\n", share);
	printf("hello2 in Kernel: %d\n", arg);
}

#define MAX_BUFFER_SIZE		(0x100)

void Hello()
{
	/* ȣ��Ʈ ���ڿ� ���� */
	char hostArg[MAX_BUFFER_SIZE] = "Before hipMemcpy";

	/* �޸� ���� �� Ȯ�� */
	printf("�޸� ���� ��\n");
	printf("hostArg: \"%s\"\n", hostArg);

	/* ����̽� ���� ���� */
	char *devArg;

	/* ����̽� �޸� �Ҵ� */
	CheckCudaError(hipMalloc((void **)&devArg, MAX_BUFFER_SIZE));

	/* ȣ��Ʈ -> ����̽� �Է� ������ ���� */
	CheckCudaError(hipMemcpy(devArg, hostArg, MAX_BUFFER_SIZE, hipMemcpyHostToDevice));

	Kernel_Hello<<<1,1>>>(devArg);

	/* ����̽� -> ȣ��Ʈ ��� ������ ���� */
	CheckCudaError(hipMemcpy(hostArg, devArg, MAX_BUFFER_SIZE, hipMemcpyDeviceToHost));

	printf("\n");

	printf("Host -> Device / Device -> Host �޸� ���� �Ϸ�...\n");

	printf("\n");

	/* ��� ��� �� ���� */
	printf("�޸� ���� ��\n");
	printf("hostArg: \"%s\"\n", hostArg);
}