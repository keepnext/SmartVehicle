#include "hip/hip_runtime.h"
#include "../common.cuh"

#include "../bmpfuncs.h"

__global__ void Kernel_Rotation(float *destData, float *srcData, int width, int height, float sinTheta, float cosTheta)
{
	/* ��, ������ ���� ���� x��, y�� �ɼ� */
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	/* �츮�� ���ϴ� �ƿ�ǲ �̹��� ���� �ɼ� */
	int offset = y * width + x;
	
	/*
	 * ���� ���� �ʾƵ� �Ǵ� ������� �����ϱ� ���� if ���� ���
	 * (�ش� ���� ���� �����常 ���� �ϵ��� ��)
	 * ȣ��Ʈ �ڵ忡�� Ŀ�� �Լ��� ���ڸ� ������ ��
	 * �̹��� ���� x ���� ������� ������ ������ ������ �����Ͽ� if �� ���� ����ϴ� ���� ����
	 */
	if (x < width && y < height) 
	{
		/* �����̼� ��� ���� */
		float x0 = width / 2.0f;
		float y0 = height / 2.0f;

		float xOffset = x - x0;
		float yOffset = y - y0;

		int xPos = (int)(xOffset * cosTheta + yOffset * sinTheta + x0);
		int yPos = (int)(yOffset * cosTheta - xOffset * sinTheta + y0);

		if ((xPos >= 0) && (xPos < width) && (yPos >= 0) && (yPos < height))
		{
			destData[offset] = srcData[yPos * width + xPos];
		}
	}
}

void Rotation()
{
	/* ȣ��Ʈ ������ �Է� */
	float theta = 3.14159f / 6.0f;
	float cosTheta = cosf(theta);
	float sinTheta = sinf(theta);
	printf("theta = %f (cos theta = %f, sin theta = %f)\n", theta, cosTheta, sinTheta);

	int imageWidth, imageHeight;
	const char *inputFile = ".\\4code\\input.bmp";
	const char *outputFile = ".\\4code\\output.bmp";

	/* �̹��� �ε� */
	float *inputImage = ReadImage(inputFile, &imageWidth, &imageHeight);
	int imageSize = imageWidth * imageHeight;
	
	/* ȣ��Ʈ �޸� �Ҵ� */
	float *outputImage = (float *)malloc(imageSize * sizeof(float));
	
	/* ȣ��Ʈ �޸� �Ҵ� ���� ó�� */
	if (outputImage == NULL)
	{
		fprintf(stderr, "ȣ��Ʈ �޸� �Ҵ� ����... ����\n");
		exit(EXIT_FAILURE);
	}

	printf("%d\n", imageWidth);
	printf("%d\n", imageHeight);
	printf("%f\n", sinTheta);
	printf("%f\n", cosTheta);

	/* ����̽� ���� ���� */
	float *devOutput = NULL;
	float *devInput = NULL;

	/* ����̽� �޸� �Ҵ� */
	CheckCudaError(hipMalloc((void **)&devOutput, imageSize * sizeof(float)));
	CheckCudaError(hipMalloc((void **)&devInput, imageSize * sizeof(float)));

	/* ȣ��Ʈ -> ����̽� �Է� ������ ���� */
	CheckCudaError(hipMemcpy(devInput, inputImage, imageSize * sizeof(float), hipMemcpyHostToDevice));

	/***********************************************************************************************************************************
	/ ��� ���� �� ��ϴ� �������� ������ �����ϴ� ����� ������ ���� �� ������ ������ �� �� �ִ�.
	/***********************************************************************************************************************************
	/ (1) ���귮�� �˰� GPU ���� �������� ���� ����
	/ �ٶ����� ����� ��� ������ ������ ���� ���̴�.
	/ 
	/ imageWidth �Ǵ� imageHeight �� 0 �� �� dimGrid �� (0, 0)
	/ imageWidth �Ǵ� imageHeight �� 1 �̻� dimBlock.x, dimBlock.y ������ �� dimGrid �� (1, 1) (��� 1 ��)
	/ imageWidth �Ǵ� imageHeight �� dimBlock.x + 1, dimBlock.y + 1 �̻� 2 * dimBlock.x, 2 * dimBlock.y ������ �� dimGrid �� (2, 2) (��� 2 x 2 ��)
	/ ...
	/
	/ �׷���, ���� ��� ���� ���𹮿���, �ܼ��� dim3 dimGrid(imageWidth / dimBlock.x, imageHeight / dimBlock.y) �� ������ ��� ������ �߻��Ѵ�.
	/ ������ "(������) / (������)" ���� ���� ���ϴ� �����̱� ������ ������ ���� ������ ������ 1 �� �۴��� �������� �� �������Ƿ� ������ ����� �ݿ����� ���Ѵ�.
	/ 
	/ ���� ���, ����ڰ� "(���� 1 �̻� imageWidth �̸�) x (���� 1 �̻� imageHeight �̸�)" ������ �����带 ����ϰ��� �� �� ����� ������ 0 �� �Ǿ������.
	/ �ᱹ ����ϰ��� �ϴ� �������� ������ imageWidth(imageHeight) �� ����� �ƴ� ��� ����� ������ ���ڶ�� �ȴ�.
	/ 
	/ ���� �̷��� �������� �����ϱ� ���� ������ ���� ó���� �� �ִ� ���� �ٶ����ϴ�.
	/ (�̷��� �ϸ� �� ������ �ʴµ��� �ұ��ϰ� �� ���� ������ �����尡 ���Ե� �� �ְ� �Ǵµ�, �̴� Ŀ�� �Լ� ������ if ������ ������ �ش�.)
	/***********************************************************************************************************************************
	/ (2) ���귮�� �𸣰ų�, ���귮�� ������ GPU ������ �ʰ��ϴ� �뷮�� ���� ����
	/ ���� ������� ���� '����'���� ��� ����� ����ڰ� ��� ���� GPU ������ �ִ� ��� ���� �� �ִ� �������� ������ �ʰ����� �ʵ���
	/ ����� ������ ��ϴ� �������� ������ ������ �׸��� ��Ȯ�� ����ϰ�, Ŀ�� �Լ� ������ while ������ �������� �۾� ������ �����ϴ� ���� ���� ����.
	/***********************************************************************************************************************************/
	
	/*
	 * �̹��� ����� �˰� �ְ�, �� �̹��� ����� GPU ����� ����� �ִ� ������ ���� �����Ƿ� (1) ��� ���
	 * ���⿡��, ������ ���� �̹����� ���ο� ���� �����ŭ ��Ȯ�� ������ �ָ� Ŀ�� �Լ� ������ if ���� ����� �ʿ䰡 ������,
	 * ���� �ִ� ��� ������ �ʰ����� ���� ��� while ���� ����� �ʿ䰡 ����.
	 * (�ִ� ��� ������ �ʰ��ϴ� �뷮 ������ ��� Ŀ�� �Լ� ������ while ���� ���� �� ������� ���� �� ���� �ϵ��� ó���ؾ� ��
	 * 
	 * ������, ������ ����: 30 x 20 / ��� ����: 20 x 20
	 */
	int xN = 30; // ��ϴ� ����� x�� �������� ����
	int yN = 20; // ��ϴ� ����� y�� �������� ����
	dim3 dimBlock(xN, yN);
	dim3 dimGrid((imageWidth + dimBlock.x - 1) / dimBlock.x, (imageHeight + dimBlock.y - 1) / dimBlock.y);
	
	/* �̹��� �����̼� Ŀ�� �Լ� ���� */
	Kernel_Rotation<<<dimGrid,dimBlock>>>(devOutput, devInput, imageWidth, imageHeight, sinTheta, cosTheta);

	/* Ŀ�� �Լ� ���� ó�� */
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "���� �ڵ�: %s ... ����\n", hipGetErrorString(err));
		if (err == hipErrorInvalidConfiguration)
		{
			printf("��ϴ� ��� ������ �ִ� ������ ������ �ʰ��߽��ϴ�!\n\
�ǽ� 2code �� 'Maximum number of threads per block' �׸��� �����Ͽ�\n\
dimBlock �� �Ķ���͸� �˸��� ������ �ٲټ���.\n");
		}
		exit(EXIT_FAILURE);
	}

	/* ����̽� -> ȣ��Ʈ ��� ������ ���� */
	CheckCudaError(hipMemcpy(outputImage, devOutput, imageSize * sizeof(float), hipMemcpyDeviceToHost));

	/* ��� �̹��� ���� */
	StoreImage(outputImage, outputFile, imageHeight, imageWidth, inputFile);

	/* ����̽� �޸� ���� */
	CheckCudaError(hipFree(devOutput));
	CheckCudaError(hipFree(devInput));

	/* ȣ��Ʈ �޸� ���� */
	free(outputImage);
}