/*
*	CUDA ����̽� ���� ȹ�� ���� �Լ� �� �ڵ� �м�
*/

////////////////////////////////////////////////////////////////////////////////

#include "../common.cuh"

// SM (Streaming Multiprocessor) �������� SM �ϳ��� �ھ� ���� ��ȯ
int ConvertSMVerToCores(int major, int minor)
{
	typedef struct {
		int SM; // 16���� 0 x (SM major version) (SM minor version)
		int Cores;
	} SM2Cores;

	SM2Cores coresPerSM[] =
	{
		/* ���÷� */
		{ 0x30, 192 },
		{ 0x32, 192 },
		{ 0x35, 192 },
		{ 0x37, 192 },

		/* �ƽ��� */
		{ 0x50, 128 },
		{ 0x52, 128 },
		{ 0x53, 128 },

		/* �Ľ�Į */
		{ 0x60, 64 },
		{ 0x61, 128 },
		{ 0x62, 128 },

		/* ��Ÿ */
		{ 0x70, 64 },

		{ -1, -1 }
	};

	int i = 0;

	while (coresPerSM[i].SM != -1)
	{
		if (coresPerSM[i].SM == ((major << 4) + minor))
		{
			return coresPerSM[i].Cores;
		}

		i++;
	}

return coresPerSM[i - 1].Cores;
}

void PrintDeviceInfo(int device)
{
	int driverVersion = 0, runtimeVersion = 0;

	// CUDA ���� �Լ� / GPU �ڿ� ����� ���� �ش� ��ȣ�� ����̽� �Ҵ�
	hipSetDevice(device);

	// CUDA ���� ����ü / ����̽� ���� ���� ����
	hipDeviceProp_t deviceProp;

	// CUDA ���� �Լ��� ���� ����̽� ���� �ε�
	hipGetDeviceProperties(&deviceProp, device);
	
	// CUDA ���� �Լ��� ���� ����̹� ����, ��Ÿ�� ���� �ε�
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);

	// CUDA ���� ����ü hipDeviceProp_t �� ����� ���� ���
	printf("Device %d: \"%s\"\n", device, deviceProp.name);
	printf("  CUDA Driver Version / Runtime Version:         %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
	printf("  CUDA Compute Capability version:               %d.%d\n", deviceProp.major, deviceProp.minor);
	
	printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
		(float)deviceProp.totalGlobalMem / (1024.0f * 1024.0f), (unsigned long long)deviceProp.totalGlobalMem);

	printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
		deviceProp.multiProcessorCount,
		ConvertSMVerToCores(deviceProp.major, deviceProp.minor),
		ConvertSMVerToCores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
	
	printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

#if CUDART_VERSION >= 5000
	// ��� ��Ÿ�� 5.0 �̻���� hipDeviceProp_t ����ü ���� ����
	printf("  Memory Clock rate:                             %.0f MHz\n", deviceProp.memoryClockRate * 1e-3f);
	printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);

	if (deviceProp.l2CacheSize)
	{
		printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
	}
#endif //CUDART_VERSION >= 5000

	printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
		deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
		deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
	printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
		deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
	printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
		deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);

	printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
	printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
	printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
	printf("  Warp size:                                     %d\n", deviceProp.warpSize);
	printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
	printf("  Max dimension size of a thread block (x,y,z):  (%d, %d, %d)\n",
		deviceProp.maxThreadsDim[0],
		deviceProp.maxThreadsDim[1],
		deviceProp.maxThreadsDim[2]);
	printf("  Max dimension size of a grid size    (x,y,z):  (%d, %d, %d)\n",
		deviceProp.maxGridSize[0],
		deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);
	printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
	printf("  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
	printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
	printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
	printf("  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
	printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
	printf("  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
	printf("  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
	printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif

	printf("  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
	printf("  Supports Cooperative Kernel Launch:            %s\n", deviceProp.cooperativeLaunch ? "Yes" : "No");
	printf("  Supports MultiDevice Co-op Kernel Launch:      %s\n", deviceProp.cooperativeMultiDeviceLaunch ? "Yes" : "No");
	printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

	const char *computeMode[] =
	{
		"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
		"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
		"Prohibited (no host thread can use ::hipSetDevice() with this device)",
		"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
		"Unknown",
		NULL
	};
	printf("  Compute Mode:\n");
	printf("     < %s >\n", computeMode[deviceProp.computeMode]);
}

void PrintCUDADeviceList()
{
	// CUDA ����̽� ���� ��������
	int deviceCount;

	// CUDA ���� ����̽��� ���� ��� 'hipErrorNoDevice' ��ȯ
	// ���� �׷��� ����̹��� ���� ��� 'hipErrorInsufficientDriver' ��ȯ
	CheckCudaError(hipGetDeviceCount(&deviceCount));

	printf("--- CUDA ���� ����̽� ����: %d\n", deviceCount);
	printf("\n");
	printf("============================================\n");

	for (int i = 0; i < deviceCount; i++)
	{
		printf("CUDA ����̽� %d��\n", i + 1);
		printf("-------------------------------------- \n");

		// �� CUDA ����̽��� ���� ��� �Լ� ȣ��
		PrintDeviceInfo(i);

		printf("-------------------------------------- \n");
	}

	printf("============================================\n");
}