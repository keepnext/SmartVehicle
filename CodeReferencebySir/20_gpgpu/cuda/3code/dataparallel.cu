#include "hip/hip_runtime.h"
/*
*	���� ������ ó�� Ŀ�� �Լ� ���� �� ȣ��Ʈ �ڵ� �ۼ�
*/

////////////////////////////////////////////////////////////////////////////////

#include "../common.cuh"

#define MEM_SIZE 10

__global__ void Kernel_DataParallel(int *A, int *B, int *C, int numElements)
{
	/*
	 * Ŀ�� ȣ��� ���� <<<����� ����,��ϴ� �������� ����>>> �� ����
	 * ��ü �����带 �Ϸķ� �� ����� (���� ��ȣ �ο�)
	 */
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	/*
	 * tid < numElements �� �ǹ�
	 * ���� �������� ������ �� �´� ������ �����带 ����ϴ� ���� ���� �����Ƿ�,
	 * �������� ������ ����� �ֵ� ���� ������ ������ŭ�� ������鸸 ������ �����ϵ��� ��
	 */
	if (tid < numElements)
	{
		C[tid] = A[tid] + B[tid];
	}

	// To do �ٸ� ������ �߰��� ������.
}

void SequentialAdd(const int *a, const int *b, int *result, int numElements)
{
	for (int i = 0; i < numElements; i++)
	{
		result[i] = a[i] + b[i];
	}
}

void DataParallel()
{
	int numElements = MEM_SIZE;

	size_t bufferSize = numElements * sizeof(int);

	/* ȣ��Ʈ �޸� �Ҵ� */
	int *A = (int *)malloc(bufferSize);
	int *B = (int *)malloc(bufferSize);
	int *C = (int *)malloc(bufferSize);
	int *CSeq = (int *)malloc(bufferSize);

	/* ȣ��Ʈ �޸� �Ҵ� ���� ó�� */
	if (A == NULL || B == NULL || C == NULL)
	{
		fprintf(stderr, "ȣ��Ʈ �޸� �Ҵ� ����... ����\n");
		exit(EXIT_FAILURE);
	}

	/* ȣ��Ʈ ������ �Է� */
	for (int i = 0; i < numElements; i++)
	{
		A[i] = i;
		B[i] = i;
		C[i] = 0;
	}

	/* ȣ��Ʈ �Է� ������ Ȯ�� */
	printf("=== A �Է� �� ===\n");
	for (int i = 0; i < numElements; i++)
	{
		printf("%d ", A[i]);
	}
	printf("\n");

	printf("\n");
	printf("=== B �Է� �� ===\n");
	for (int i = 0; i < numElements; i++)
	{
		printf("%d ", B[i]);
	}
	printf("\n");

	/* ������ �� */
	SequentialAdd(A, B, CSeq, numElements);

	/* ������ ��� ��� */
	printf("\n");
	printf("=== ������ ��� ��� ===\n");
	for (int i = 0; i < numElements; i++)
	{
		printf("%d ", CSeq[i]);
	}
	printf("\n");

	//////////////////////////////////////////////////////////////////
	
	printf("\n");
	printf("------------------- ���� --------------------\n");

	/* ����̽� ���� ���� */
	int *devA = NULL;
	int *devB = NULL;
	int *devC = NULL;

	/* ����̽� �޸� �Ҵ� */
	CheckCudaError(hipMalloc((void **)&devA, bufferSize));
	CheckCudaError(hipMalloc((void **)&devB, bufferSize));
	CheckCudaError(hipMalloc((void **)&devC, bufferSize));

	/* ȣ��Ʈ -> ����̽� �Է� ������ ���� */
	CheckCudaError(hipMemcpy(devA, A, bufferSize, hipMemcpyHostToDevice));
	CheckCudaError(hipMemcpy(devB, B, bufferSize, hipMemcpyHostToDevice));

	/* ������ ���� ���� Ŀ�� �Լ� ���� */
	int threadsPerBlock = 5;
	int blocksPerGrid = numElements;
	Kernel_DataParallel<<<blocksPerGrid,threadsPerBlock>>>(devA, devB, devC, numElements);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "���� �ڵ�: %s ... ����\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	/* ����̽� -> ȣ��Ʈ ��� ������ ���� */
	CheckCudaError(hipMemcpy(C, devC, bufferSize, hipMemcpyDeviceToHost));

	/* ��� ��� */
	printf("\n");
	for (int i = 0; i < numElements; i++)
	{
		printf("%d ", C[i]);
	}
	printf("\n");

	/* ����̽� �޸� ���� */
	CheckCudaError(hipFree(devA));
	CheckCudaError(hipFree(devB));
	CheckCudaError(hipFree(devC));

	/* ȣ��Ʈ �޸� ���� */
	free(A);
	free(B);
	free(C);
	free(CSeq);

	printf("\n");
	printf("----------------- ���� ���� -----------------\n");
	printf("\n");
}